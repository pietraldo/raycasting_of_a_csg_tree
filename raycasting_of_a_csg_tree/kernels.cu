#include "hip/hip_runtime.h"
#include "kernels.cuh"

__host__ __device__ void MultiplyVectorByMatrix4(float* vector, float* matrix)
{
	float result[4] = { 0 };
	for (int i = 0; i < 4; i++) {
		result[i] = 0;
		for (int j = 0; j < 4; j++) {
			result[i] += vector[j] * matrix[i * 4 + j];
		}
	}
	for (int i = 0; i < 4; i++) {
		vector[i] = result[i];
	}
}

__host__ __device__ void NormalizeVector4(float* vector)
{
	float length = sqrt(vector[0] * vector[0] + vector[1] * vector[1] + vector[2] * vector[2] + vector[3] * vector[3]);
	vector[0] /= length;
	vector[1] /= length;
	vector[2] /= length;
	vector[3] /= length;
}

__host__ __device__ void NormalizeVector3(float* vector)
{
	float length = sqrt(vector[0] * vector[0] + vector[1] * vector[1] + vector[2] * vector[2]);
	vector[0] /= length;
	vector[1] /= length;
	vector[2] /= length;
}

__host__ __device__ bool TreeContains(Node* tree, float x, float y, float z, int nodeIndex)
{

	if (tree[nodeIndex].left == -1 && tree[nodeIndex].right == -1)
	{
		return SphereContains(tree[nodeIndex].x, tree[nodeIndex].y, tree[nodeIndex].z, tree[nodeIndex].radius, x, y, z);
	}
	else
	{
		bool left = TreeContains(tree, x, y, z, tree[nodeIndex].left);
		bool right = TreeContains(tree, x, y, z, tree[nodeIndex].right);
		if (tree[nodeIndex].operation == 0)
			return SphereSubstraction(left, right);
		else if (tree[nodeIndex].operation == 1)
			return SphereIntersection(left, right);
		else
			return SphereUnion(left, right);
		//return tree[nodeIndex].functionPtr(left, right);
	}
}
__device__ bool BlockingLightRay(DevSphere* spheres, size_t sphere_count, float* pixelPosition, float* lightRay, Node* dev_tree)
{

	pixelPosition[0] += 0.001 * lightRay[0];
	pixelPosition[1] += 0.001 * lightRay[1];
	pixelPosition[2] += 0.001 * lightRay[2];
	for (int k = 0; k < sphere_count; k++)
	{
		float t1, t2;
		if (!IntersectionPoint(&spheres[k], pixelPosition, lightRay, t1, t2)) continue;

		float intersection1[3];
		for (int i = 0; i < 3; i++)
			intersection1[i] = pixelPosition[i] + (t1 + 0.001) * lightRay[i];

		if (t1 > 0 && TreeContains(dev_tree, intersection1[0], intersection1[1], intersection1[2], 0))
		{
			return true;
		}

		float intersection2[3];
		for (int i = 0; i < 3; i++)
			intersection2[i] = pixelPosition[i] + (t2 - 0.001) * lightRay[i];

		if (t2 > 0 && TreeContains(dev_tree, intersection2[0], intersection2[1], intersection2[2], 0))
		{
			return true;
		}
	}
	return false;
}

__global__ void child()
{
	int i = threadIdx.x;
	//printf("Hello from child\n");
}

__global__ void GoTree(Node* arr, float3 point, size_t sphere_count, bool* result)
{
	__shared__ bool results[128];
	//printf("Hello from GoTree\n");
	int index = threadIdx.x + sphere_count - 1;
	if (index >= 2 * sphere_count - 1)
		return;


	// first is a leaf
	results[index] = SphereContains(arr[index].x, arr[index].y, arr[index].z, arr[index].radius, point.x, point.y, point.z);
	__syncthreads();
	//printf("index %d:  %d\n", index, results[index]);

	int prev = index;
	index = arr[index].parent;

	while (index != -1)
	{

		if (arr[index].right == prev) return;

		if (arr[index].operation == 0)
			results[index] = SphereSubstraction(results[arr[index].right], results[arr[index].left]);
		else if (arr[index].operation == 1)
			results[index] = SphereIntersection(results[arr[index].right], results[arr[index].left]);
		else
			results[index] = SphereUnion(results[arr[index].right], results[arr[index].left]);
		__syncthreads();
		//printf("index %d:  %d\n", index, results[index]);

		if (index == 0)
		{
			/**result = results[index];*/
			return;
		}

		prev = index;
		index = arr[index].parent;
	}

}
__global__ void CalculateInterscetion(unsigned char* dev_texture_data, int width, int height, DevSphere* spheres, size_t sphere_count,
	float* pojection, float* view, float* camera_pos, float* light_pos, Node* dev_tree, float* dev_intersecion_points, float* dev_intersection_result)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x >= width || y >= height)
		return;

	int iindex = (x + y * width) * sphere_count * 2;
	float min = 10000;
	for (int i = 0; i < sphere_count; i++)
	{
		if (x == 400 && y == 300)
		{
			printf("t1: %f, t2: %f\n", dev_intersecion_points[iindex + 2 * i], dev_intersecion_points[iindex + 2 * i + 1]);
		}
		if (dev_intersecion_points[iindex + 2 * i] < min && dev_intersecion_points[iindex+2*i]>0)
			min = dev_intersecion_points[iindex + 2 * i];
	}
		
	if (x == 400 && y == 300)
	{
		printf("min: %f\n", min);
	}
	dev_intersection_result[x + y * width] = min;
}


__global__ void RayWithSphereIntersectionPoints(unsigned char* dev_texture_data, int width, int height, DevSphere* spheres, size_t sphere_count,
	float* projection, float* view, float* camera_pos, float* light_pos, Node* dev_tree, float* dev_intersecion_points)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= width || y >= height)
		return;

	float stepX = 2 / (float)width;
	float stepY = 2 / (float)height;

	float ray[3] = { -1 + x * stepX, -1 + y * stepY, 1.0f };
	float target[4] = { ray[0], ray[1], ray[2], 1.0f };

	MultiplyVectorByMatrix4(target, projection);
	for (int i = 0; i < 4; i++)
		target[i] /= target[3];
	NormalizeVector4(target);
	target[3] = 0.0f;

	MultiplyVectorByMatrix4(target, view);

	ray[0] = target[0];
	ray[1] = target[1];
	ray[2] = target[2];

	int index = (x + y * width) * sphere_count * 2;
	for (int k = 0; k < sphere_count; k++)
	{
		float t1=-1, t2=-1;
		IntersectionPoint(&spheres[k], camera_pos, ray, t1, t2);

		dev_intersecion_points[index + 2 * k] = t1;
		dev_intersecion_points[index + 2 * k + 1] = t2;
		if (x == 400 && y == 300)
		{
			printf("t1: %f, t2: %f\n", t1, t2);
		}
	}
}

void UpdateOnGPU(unsigned char* dev_texture_data, int width, int height, DevSphere* devSpheres,
	size_t sphere_count, float* projection, float* view, float* camera_pos, float* light_pos, Node* dev_tree, float* dev_intersecion_points, float* dev_intersection_result)
{
	dim3 block(16, 16);
	dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

	

	RayWithSphereIntersectionPoints << <grid, block >> > (dev_texture_data, width, height, devSpheres, sphere_count, projection, view, camera_pos, light_pos, dev_tree, dev_intersecion_points);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("RayWithSphereIntersectionPoints launch error: %s\n", hipGetErrorString(err));
	}
	hipDeviceSynchronize();

	CalculateInterscetion << <grid, block >> > (dev_texture_data, width, height, devSpheres, sphere_count, projection, view, camera_pos, light_pos, dev_tree, dev_intersecion_points, dev_intersection_result);
	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("CalculateInterscetion launch error: %s\n", hipGetErrorString(err));
	}
	hipDeviceSynchronize();

	ColorPixel << <grid, block >> > (dev_texture_data, width, height, devSpheres, sphere_count, projection, view, camera_pos, light_pos, dev_tree, dev_intersecion_points, dev_intersection_result);
	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("CalculateInterscetion launch error: %s\n", hipGetErrorString(err));
	}
	hipDeviceSynchronize();

}

__global__ void ColorPixel(unsigned char* dev_texture_data, int width, int height, DevSphere* spheres, size_t sphere_count,
	float* pojection, float* view, float* camera_pos, float* light_pos, Node* dev_tree, float* dev_intersecion_points, float* dev_intersection_result)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= width || y >= height)
		return;

	float colorf = dev_intersection_result[x + y * width];
	if (x == 400 && y == 300)
	{
		printf("colorf: %f\n", colorf);
	}

	unsigned char color = (colorf < 100 & colorf>0) ? 255 : 0;

	int index = 3 * (y * width + x);
	dev_texture_data[index] = color;
	dev_texture_data[index + 1] = color;
	dev_texture_data[index + 2] = color;
}


__host__ __device__ bool IntersectionPoint(DevSphere* sphere, float* rayOrigin, float* rayDirection, float& t1, float& t2)
{
	float a = dot3(rayDirection, rayDirection);
	float rayMinusSphere[3] = { rayOrigin[0] - sphere->position[0], rayOrigin[1] - sphere->position[1], rayOrigin[2] - sphere->position[2] };
	float b = 2 * dot3(rayDirection, rayMinusSphere);
	float c = dot3(rayMinusSphere, rayMinusSphere) - sphere->radius * sphere->radius;

	float discriminant = b * b - 4 * a * c;
	if (discriminant < 0)
	{
		return false;
	}
	t1 = (-b - sqrt(discriminant)) / (2 * a);
	t2 = (-b + sqrt(discriminant)) / (2 * a);
	return true;
}

__host__ __device__ float dot3(float* a, float* b)
{
	return a[0] * b[0] + a[1] * b[1] + a[2] * b[2];
}

__host__ __device__ bool SphereSubstraction(bool a, bool b)
{
	return a && !b;
}

__host__ __device__ bool SphereIntersection(bool a, bool b)
{
	return a && b;
}

__host__ __device__ bool SphereUnion(bool a, bool b)
{
	return a || b;
}

__host__ __device__ bool SphereContains(float sx, float sy, float sz, float sr, float x, float y, float z)
{
	return (x - sx) * (x - sx) + (y - sy) * (y - sy) + (z - sz) * (z - sz) < sr * sr;
}